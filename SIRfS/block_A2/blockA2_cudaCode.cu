#include "hip/hip_runtime.h"
#include "../templates/matrix2D.h"
#include "kernels_wrappers.h"
#include <time.h>

#define D 5
#define PATCH_DIM 11
#define PATCH_BLOCK_DIM 64

__global__ void conv2mat_kernel(int* device_i0, int* device_j0, int oi, int oj, int maskRows, int maskCols, int* device_idx)
{
	int global_index = threadIdx.x + blockIdx.x*blockDim.x;
	int maskLinearSize = maskRows*maskCols;
	//device_counter_dim = 0;
	
	__shared__ int i[THREADS_PER_ONE_DIM_BLOCK], j[THREADS_PER_ONE_DIM_BLOCK], keep[THREADS_PER_ONE_DIM_BLOCK];
	
	if(global_index < maskLinearSize)
	{
		i[threadIdx.x] = device_i0[global_index] + oi;
		j[threadIdx.x] = device_j0[global_index] + oj;
		
		keep[threadIdx.x] = (i[threadIdx.x] < maskRows) & (j[threadIdx.x] < maskCols);
		
		if(keep[threadIdx.x])
		{
			//If the updated indeces are within the mask matrix dimensions range, linearize them and store them in a vector (idx)
			// Matlab's sub2ind  loops firstly on each row, then on columns thus the linearization below (reads column by column, not line by line)
			device_idx[global_index] = i[threadIdx.x] + maskRows * j[threadIdx.x];
		}
		else
		{
			//if the updated indeces are out of range, store -1 as linear index and increase counter
			device_idx[global_index] = -1;
		}
	}
}

hipError_t launch_conv2mat_kernel(int** i0, int** j0, int oi, int oj, int maskRows, int maskCols, int** idx)
{
	int *device_i0, *device_j0, *device_idx;
	int mask_matrix_linear_size = maskRows*maskCols;
	
	//allocate GPU memory for inputs
	hipMalloc((void**)&device_i0, sizeof(int) * mask_matrix_linear_size);
	hipMalloc((void**)&device_j0, sizeof(int) * mask_matrix_linear_size);
	
	//allocate GPU memory for outputs
	hipMalloc((void**)&device_idx, sizeof(int) * mask_matrix_linear_size);
	
	//transfer data to GPU RAM
	hipMemcpy(device_i0, *i0, sizeof(int) * mask_matrix_linear_size, hipMemcpyHostToDevice);
	hipMemcpy(device_j0, *j0, sizeof(int) * mask_matrix_linear_size, hipMemcpyHostToDevice);
	
	//launch kernel
	dim3 dimBlock(64);
	dim3 dimGrid((mask_matrix_linear_size + dimBlock.x -1)/dimBlock.x);
	
	conv2mat_kernel <<<dimGrid, dimBlock>>> (device_i0, device_j0, oi, oj, maskRows, maskCols, device_idx);
	
	//copy the results from GPU RAM into CPU RAM
	hipMemcpy(*idx, device_idx, sizeof(int) * mask_matrix_linear_size, hipMemcpyDeviceToHost);
	
	hipFree(device_i0);
	hipFree(device_j0);
	hipFree(device_idx);
	
	hipDeviceSynchronize();
	return hipGetLastError();
}

__global__ void getBorderNormals_computePatch(bool *mask, double *P, int* device_d_vector, bool* device_patch, int *counter, int P_rows, int mask_rows, int mask_cols)
{
	int global_index = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(global_index < P_rows)
	{
		int mask_x[PATCH_DIM], mask_y[PATCH_DIM];
		
		//compute indeces of mask
		for(int j = 0; j < PATCH_DIM; j++)
		{
			mask_x[j] = device_d_vector[j] + P[2*global_index];
			mask_y[j] = device_d_vector[j] + P[2*global_index + 1];	
		}
		
		for(int i = 0; i < PATCH_DIM; i++)
        {
			for(int j = 0; j < PATCH_DIM; j++)
			{
				device_patch[global_index*PATCH_DIM*PATCH_DIM + i*PATCH_DIM +j]  = mask[ mask_x[i]*mask_cols + mask_y[j] ];
            
				if(mask[ mask_x[i]*mask_cols + mask_y[j] ] != 0)
					counter[global_index]++;

			}
        }
    }
}

hipError_t launch_getBorderNormals_computePatch(Matrix2D<bool> &mask, Matrix2D<double> &P, int **d_vector, std::vector<Matrix2D<bool> > &patch_vector, int **counter, int P_rows, int mask_rows, int mask_cols)
{
	bool *device_mask, *device_patch;
	double *device_P;
	int *device_d_vector;
	int *device_counter;
	int d = 5;

	//allocate GPU memory for inputs
	hipHostAlloc((void**)&device_mask, sizeof(bool) * mask.getDim(), hipHostMallocDefault);
    hipHostAlloc((void**)&device_P, sizeof(double) * P.getDim(), hipHostMallocDefault);
    hipHostAlloc((void**)&device_d_vector, sizeof(int) * (2*d + 1), hipHostMallocDefault);
    hipHostAlloc((void**)&device_counter, sizeof(int) * P.getRows(), hipHostMallocDefault);
    //allocate GPU memory for output
    hipHostAlloc((void**)&device_patch, sizeof(bool) * (2*d + 1)*(2*d + 1)*P.getRows(), hipHostMallocDefault);
    
    //transfer data to GPU RAM
    hipMemcpy(device_P, P.getContainerPointer(), sizeof(double) * P.getDim(), hipMemcpyHostToDevice);
    hipMemcpy(device_mask, mask.getContainerPointer(), sizeof(bool) * mask.getDim(), hipMemcpyHostToDevice);
    hipMemcpy(device_d_vector, *d_vector, sizeof(int) * (2*d + 1), hipMemcpyHostToDevice);
    hipMemcpy(device_counter, *counter, sizeof(int) * P.getRows(), hipMemcpyHostToDevice);
    
    //launch kernel
    dim3 dimBlock(64);
	dim3 dimGrid((P.getRows()+dimBlock.x-1)/dimBlock.x);

	getBorderNormals_computePatch <<<dimGrid, dimBlock>>> (device_mask, device_P, device_d_vector, device_patch, device_counter, P.getRows(), mask.getRows(), mask.getCols());
	
	//copy the results from GPU RAM into CPU RAM
    for(int i = 0; i < P.getRows(); i++)
		hipMemcpy(patch_vector[i].getContainerPointer(), device_patch + i*(2*d + 1)*(2*d + 1), sizeof(bool)*(2*d + 1)*(2*d + 1), hipMemcpyDeviceToHost);
	hipMemcpy(*counter, device_counter, sizeof(int) * P.getRows(), hipMemcpyDeviceToHost);
	
	//free GPU memory
	hipHostFree(device_mask);
	hipHostFree(device_patch);
	hipHostFree(device_d_vector);
	hipHostFree(device_P);
	
	hipDeviceSynchronize();
	return hipGetLastError();
}

__global__ void borderNormals_compute_temp_a(int* ii, int *jj, int dim, bool *temp_a, int* substract_ii, int *substract_jj)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(row < dim && col < dim)
	{
		substract_ii[row*dim +col] =  ii[row] - ii[col];
		substract_jj[row*dim +col] =  jj[row] - jj[col];
		
		__syncthreads();
		substract_ii[row*dim +col] = substract_ii[row*dim +col]*substract_ii[row*dim +col];
		substract_jj[row*dim +col] = substract_jj[row*dim +col]*substract_jj[row*dim +col];
		
		__syncthreads();
		substract_ii[row*dim +col] = substract_ii[row*dim +col] + substract_jj[row*dim +col];
		
		__syncthreads();
		if(substract_ii[row*dim +col] <= 2)
		{
			temp_a[row*dim +col] = 1;
		}
		else
		{
			temp_a[row*dim +col] = 0;
		}
	}
}

hipError_t launch_borderNormals_compute_temp_a(int **ii, int **jj, int dim, Matrix2D<bool>& temp_a)
{
	bool *device_temp_a;
	int *device_ii, *device_jj, *substract_ii, *substract_jj;
	Matrix2D<int> sub_ii(dim, dim);
	
	//allocate GPU memory for inputs
	hipMalloc((void**)&device_ii, sizeof(int) * dim);
	hipMalloc((void**)&device_jj, sizeof(int) * dim);
	hipMalloc((void**)&substract_ii, sizeof(int) * dim*dim);
	hipMalloc((void**)&substract_jj, sizeof(int) * dim*dim);
	
	//allocate GPU memory for output
	hipMalloc((void**)&device_temp_a, sizeof(bool) * temp_a.getDim());
	
	//transfer data to GPU RAM
	hipMemcpy(device_ii, *ii, sizeof(int) * dim, hipMemcpyHostToDevice);
	hipMemcpy(device_jj, *jj, sizeof(int) * dim, hipMemcpyHostToDevice);
	
	//launch kernel
	dim3 dimBlock(8, 8);
	dim3 dimGrid((dim + dimBlock.x -1)/dimBlock.x, (dim + dimBlock.y -1)/dimBlock.y);
	
	
	borderNormals_compute_temp_a <<<dimGrid, dimBlock>>> (device_ii, device_jj, dim, device_temp_a, substract_ii, substract_jj);
	
	
	//copy the results from GPU RAM into CPU RAM
	hipMemcpy(temp_a.getContainerPointer(), device_temp_a, sizeof(bool) * temp_a.getDim(), hipMemcpyDeviceToHost);
	
	hipFree(device_temp_a);
	hipFree(device_ii);
	hipFree(device_jj);
	hipFree(substract_ii);
	hipFree(substract_jj);
	
	hipDeviceSynchronize();
	return hipGetLastError();
}

__global__ void initCounter(int *result) 
{
	*result = 0;
}

__global__ void borderNormals_compute_maskedP(double *P, int P_rows, double *device_P_plus, double *device_P_minus, bool *device_P_plus_mask, bool *device_P_minus_mask, bool *device_allNonZeroLines, \
int *device_notzero_lines_in_P, int mask_rows, int mask_cols)
{
	int global_index = threadIdx.x + blockIdx.x*blockDim.x;
	int d = 5;
	
	if(global_index < P_rows*2)
	{
		device_P_plus[global_index] = P[global_index]+d;
		device_P_minus[global_index] = P[global_index]-d;
		
		syncthreads();
		
		//take each column
		if(global_index %2 == 0)
		{
			if(device_P_plus[global_index] < mask_rows)
			{
				device_P_plus_mask[global_index] = 1;
			}
			else
			{
				device_P_plus_mask[global_index] = 0;
			}
		}
		else
		{
			if(device_P_plus[global_index] < mask_cols)
			{
				device_P_plus_mask[global_index] = 1;
			}
			else
			{
				device_P_plus_mask[global_index] = 0;
			}
		}
		
		if(device_P_minus[global_index] >= 1)
		{
			device_P_minus_mask[global_index] = 1;
		}
		else
		{
			device_P_minus_mask[global_index] = 0;
		}
		
		syncthreads();
		
		device_P_plus_mask[global_index] = device_P_plus_mask[global_index] & device_P_minus_mask[global_index];
		
		__syncthreads();
		
		if(global_index %2 == 0)
		{
			if(device_P_plus_mask[global_index] == 0 || device_P_plus_mask[global_index+1] == 0)
			{
				device_allNonZeroLines[global_index/2] = 0;
			}
			else
			{
				device_allNonZeroLines[global_index/2] = 1;
				atomicAdd(device_notzero_lines_in_P, 1);	
			}
		}
		
		__syncthreads();
		
		//printf("%d  ", device_notzero_lines_in_P);
	}

}

hipError_t launch_borderNormals_compute_maskedP(Matrix2D<double>& P, bool *allNonZeroLines, int *notzero_lines_in_P, int mask_rows, int mask_cols)
{
	hipError_t code;
	double *device_P, *device_P_plus, *device_P_minus, *host_P, *host_P_plus, *host_P_minus;
	bool *device_P_plus_mask, *device_P_minus_mask, *device_allNonZeroLines, *host_P_plus_mask, *host_P_minus_mask;
	int *device_notzero_lines_in_P;
	
	//allocate GPU memory for inputs
	code =hipHostAlloc((void**)&host_P, sizeof(double) * P.getRows()*2, hipHostMallocMapped);
	if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
	code =hipHostAlloc((void**)&host_P_plus, sizeof(double) * P.getRows()*2, hipHostMallocMapped);
	if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
    code =hipHostAlloc((void**)&host_P_minus, sizeof(double) * P.getRows()*2, hipHostMallocMapped);
    if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
    code =hipHostAlloc((void**)&host_P_plus_mask, sizeof(bool) * P.getRows()*2, hipHostMallocMapped);
    if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
    code =hipHostAlloc((void**)&host_P_minus_mask, sizeof(bool) * P.getRows()*2, hipHostMallocMapped);
    if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
    
    //allocate GPU memory for outputs
    code =hipHostAlloc((void**)&device_notzero_lines_in_P, sizeof(int), hipHostMallocDefault);
    if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);

	//transfer data to GPU RAM: 
	hipMemcpy(host_P, P.getContainerPointer(), sizeof(double) * P.getRows()*2, hipMemcpyHostToDevice);
	//for(int i = 0; i < P.getRows()*2; i++)
		//host_P[i] = P.getMatrixValue(i);
	
	// get device pointer
	code = hipHostGetDevicePointer(&device_P, host_P, 0);
	if(code != hipSuccess)
	printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
	
	code = hipHostGetDevicePointer(&device_P_plus, host_P_plus, 0);
	if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
	
	code = hipHostGetDevicePointer(&device_P_minus, host_P_minus, 0);
	if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
	
	code = hipHostGetDevicePointer(&device_P_plus_mask, host_P_plus_mask, 0);
	if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
	
	code = hipHostGetDevicePointer(&device_P_minus_mask, host_P_minus_mask, 0);
	if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
	
	code = hipHostGetDevicePointer(&device_allNonZeroLines, allNonZeroLines, 0);
	if(code != hipSuccess)
		printf("GPUassert: %s %s %d\n", hipGetErrorString(code), __FILE__, __LINE__);
	
	//launch kernel
    dim3 dimBlock(64);
	dim3 dimGrid((2*P.getRows()+dimBlock.x-1)/dimBlock.x);

	initCounter<<<1,1>>>(device_notzero_lines_in_P);
	borderNormals_compute_maskedP <<<dimGrid, dimBlock>>> (device_P, P.getRows(), device_P_plus, device_P_minus, device_P_plus_mask, device_P_minus_mask, device_allNonZeroLines, device_notzero_lines_in_P, mask_rows, mask_cols);
	
	hipDeviceSynchronize();
	
	//hipMemcpy(*allNonZeroLines, device_allNonZeroLines, sizeof(bool) * P.getRows(), hipMemcpyDeviceToHost);
	hipMemcpy(notzero_lines_in_P, device_notzero_lines_in_P, sizeof(int), hipMemcpyDeviceToHost);
	
	std::cout<<*notzero_lines_in_P<<std::endl;

	//free GPU memory
	/*hipHostFree(device_P);
	hipHostFree(device_P_plus);
	hipHostFree(device_P_minus);
	hipHostFree(device_P_plus_mask);
	hipHostFree(device_P_minus_mask);
	hipHostFree(device_allNonZeroLines);
	hipHostFree(device_notzero_lines_in_P);
	*/
	
	hipHostFree(host_P);
	hipHostFree(host_P_plus);
	hipHostFree(host_P_minus);
	hipHostFree(host_P_plus_mask);
	hipHostFree(host_P_minus_mask);
	hipHostFree(device_notzero_lines_in_P);
	

	hipDeviceSynchronize();
	return hipGetLastError();
}
