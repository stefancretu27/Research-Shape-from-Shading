#include "hip/hip_runtime.h"
#include "matrix2D.h"
//#include "matrix2D_cuda.h"

#define BLOCK_SIZE 16

template<class Type>
__global__ void matrix_multiply(Type* device_A, Type* device_B, Type* device_C, int rows_A, int rows_B, int cols_B)
{
	int y = threadIdx.y;
	int x = threadIdx.x;

	int row = y + blockIdx.y * blockDim.y;
	int col = x + blockIdx.x * blockDim.x;
	
	int cols_A = rows_B;
	//accumulator
	Type accu = (Type) 0;

	
	//shared memory where each block of threads loads submatrices of A and B
	__shared__ Type shared_A[BLOCK_SIZE][BLOCK_SIZE];
	__shared__ Type shared_B[BLOCK_SIZE][BLOCK_SIZE];
	
	int block_row, block_col, block_index;
	
	for(int index = 0; index < (cols_A + BLOCK_SIZE -1)/BLOCK_SIZE; index++)
	{
		//don't exceed matrix dimensions
		if(row < rows_A && (index*BLOCK_SIZE + x) < cols_A)
		{
			//identify the line of blocks
			block_row = row * cols_A;
			//identify block in the current line of blocks
			block_index = index*BLOCK_SIZE;
			
			//load current element in current block
			shared_A[y][x] = device_A[block_row + block_index + x];
		}
		else
		{
			shared_A[y][x] = 0.0;
		}
		
		//don't exceed matrix dimensions
		if(col < cols_B && (index*BLOCK_SIZE +y) < rows_B)
		{
			//indetify the column of blocks
			block_col = col;
			//identify block in current column of blocks
			block_index = index*BLOCK_SIZE*cols_B;
			//load element from the current block
			
			shared_B[y][x] = device_B[block_col + block_index + y*cols_B];
		}
		else
		{
			shared_B[y][x] = 0.0;
		}
		
		__syncthreads();
		
		//multiply the loaded blocks
		for(int j = 0; j < BLOCK_SIZE; j++)
			accu += shared_A[y][j]*shared_B[j][x];
			
		__syncthreads();
	}
	
	if(row < rows_A && col < cols_B)
		device_C[row*cols_B + col] = accu;
}

template<class Type> hipError_t operator_multiply(Type* device_A, Type* device_B, Type *result, dim3 grid_dim, dim3 block_dim, int A_rows, int B_rows, int B_cols)
{
	matrix_multiply <<<grid_dim, block_dim>>> (device_A, device_B, result, A_rows, B_rows, B_cols);
	
	hipDeviceSynchronize();
	return hipGetLastError();
}
//specializations
template hipError_t operator_multiply<double>(double* device_A, double* device_B, double *result, dim3 grid_dim, dim3 block_dim, int A_rows, int B_rows, int B_cols);
template hipError_t operator_multiply<float>(float* device_A, float* device_B, float *result, dim3 grid_dim, dim3 block_dim, int A_rows, int B_rows, int B_cols);
template hipError_t operator_multiply<int>(int* device_A, int* device_B, int *result, dim3 grid_dim, dim3 block_dim, int A_rows, int B_rows, int B_cols);

/* convolution */
template<class Type>
__global__ void conv2D_full(Type* input, Type* kernel, Type* output, int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols)
{
	int col = threadIdx.x + blockIdx.x*blockDim.x;
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int m, n, sourceIdx, sourceIdy;
	
	//find kernel center
	int kCenterX = (kernel_cols + 1)/2;
	int kCenterY = (kernel_rows + 1)/2;
	
	//initialize output value to 0, to accumulate to it
	output[row*output_cols + col] = 0;
	
	if(col < output_cols && row < output_rows)
	{
		for(m = 0; m < kernel_rows; m++)
		{
			for(n = 0; n < kernel_cols; n++)
			{
				sourceIdx = row + m - kCenterY;
				sourceIdy = col + n - kCenterX;

				if( sourceIdx < input_rows && sourceIdx >= 0 && sourceIdy < input_cols && sourceIdy  >= 0)
				{
					output[row*output_cols + col] += input[sourceIdx*input_cols + sourceIdy]*kernel[m*kernel_cols + n];
				}
			}
		}
	}
}

template<class Type> hipError_t conv2D_full_kernel_launcher(Type* input, Type* kernel, Type* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols)
{
	
	conv2D_full <<<grid_dim, block_dim>>> (input, kernel, output, input_rows, input_cols, kernel_rows, kernel_cols, output_rows, output_cols);
	
	hipDeviceSynchronize();
	return hipGetLastError();
}
//specializations
template hipError_t conv2D_full_kernel_launcher<double>(double* input, double* kernel, double* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);
template hipError_t conv2D_full_kernel_launcher<float>(float* input, float* kernel, float* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);
template hipError_t conv2D_full_kernel_launcher<int>(int* input, int* kernel, int* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);

template<class Type>
__global__ void conv2D_valid(Type* input, Type* kernel, Type* output, int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols)
{
	int col = threadIdx.x + blockIdx.x*blockDim.x;
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int m, n, in_row, in_col;
	
	//initialize output value to 0, to accumulate to it
	output[row*output_cols + col] = 0;
	
	if(col < output_cols && row < output_rows)
	{
		for(m = 0; m < kernel_rows; m++)
		{
			for(n = 0; n < kernel_cols; n++)
			{
				in_row = row + m;
				in_col = col + n;
				if( in_row < input_rows && in_col < input_cols)
				{
					output[row*output_cols + col] += input[in_row*input_cols + in_col] * kernel[m*kernel_cols + n];
				}
			}
		}
	}
}


template<class Type> hipError_t conv2D_valid_kernel_launcher(Type* input, Type* kernel, Type* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols)
{
	
	conv2D_valid <<<grid_dim, block_dim>>> (input, kernel, output, input_rows, input_cols, kernel_rows, kernel_cols, output_rows, output_cols);
	
	hipDeviceSynchronize();
	return hipGetLastError();
}
//specializations
template hipError_t conv2D_valid_kernel_launcher<double>(double* input, double* kernel, double* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);
template hipError_t conv2D_valid_kernel_launcher<float>(float* input, float* kernel, float* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);
template hipError_t conv2D_valid_kernel_launcher<int>(int* input, int* kernel, int* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);
	
template<class Type>
__global__ void conv2D_same(Type* input, Type* kernel, Type* output, int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols)
{
	int col = threadIdx.x + blockIdx.x*blockDim.x;
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int m, n, sourceIdx, sourceIdy;
	
	//find kernel center
	int kCenterX = (kernel_cols - 1)/2;
	int kCenterY = (kernel_rows - 1)/2;
	
	//initialize output value to 0, to accumulate to it
	output[row*output_cols + col] = (Type) 0;

	if(col < output_cols && row < output_rows)
	{
		for(m = 0; m < kernel_rows; m++)
		{
			for(n = 0; n < kernel_cols; n++)
			{
				sourceIdx = row + m - kCenterY;
				sourceIdy = col + n - kCenterX;

				if( sourceIdx < input_rows && sourceIdx >= 0 && sourceIdy < input_cols && sourceIdy >= 0)
				{
					output[row*output_cols + col] += input[sourceIdx*input_cols + sourceIdy]*kernel[m*kernel_cols + n];
				}
			}
		}
	}
}
														
template<class Type> hipError_t conv2D_same_kernel_launcher(Type* input, Type* kernel, Type* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols)
{
	
	conv2D_same <<<grid_dim, block_dim>>> (input, kernel, output, input_rows, input_cols, kernel_rows, kernel_cols, output_rows, output_cols);
	
	hipDeviceSynchronize();
	return hipGetLastError();
}
//specializations
template hipError_t conv2D_same_kernel_launcher<double>(double* input, double* kernel, double* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);
template hipError_t conv2D_same_kernel_launcher<float>(float* input, float* kernel, float* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);
template hipError_t conv2D_same_kernel_launcher<int>(int* input, int* kernel, int* output, dim3 grid_dim, dim3 block_dim, \
															int input_rows, int input_cols, int kernel_rows, int kernel_cols, int output_rows, int output_cols);
